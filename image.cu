#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <opencv2/opencv.hpp>
#include <cfloat>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda/border_interpolate.hpp>
#include <opencv2/core/cuda/vec_traits.hpp>
#include <opencv2/core/cuda/vec_math.hpp>

#include "hip/hip_vector_types.h"

__global__ void process(const cv::cuda::PtrStep<uchar3> src, cv::cuda::PtrStep<uchar3> dst, int rows, int cols, int kernelSize) {
    const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

    uint3 sum = {0,0,0};
    for (int j=-kernelSize/2; j<kernelSize/2+kernelSize%2; j++) {
        for (int i=-kernelSize/2; i<kernelSize/2+kernelSize%2; i++) {
            char2 coord = {dst_x+i, dst_y+j};
            if (coord.x < 0) coord.x = 0;
            if (coord.y < 0) coord.y = 0;
            if (coord.x >= cols) coord.x = cols-1;
            if (coord.y >= rows) coord.y = rows-1;
            uchar3 val = src(coord.y, coord.x);
            sum.x += val.x;
            sum.y += val.y;
            sum.z += val.z;
        }
    }
    sum.x /= kernelSize * kernelSize;
    sum.y /= kernelSize * kernelSize;
    sum.z /= kernelSize * kernelSize;

    dst(dst_y, dst_x).x = sum.x;
    dst(dst_y, dst_x).y = sum.y;
    dst(dst_y, dst_x).z = sum.z;
}

int divUp(int a, int b) {
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void startCUDA (cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst) {
    const dim3 block(32, 8);
    const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

    const int kernelSize = 5;

    process<<<grid, block>>>(src, dst, dst.rows, dst.cols, kernelSize);
}